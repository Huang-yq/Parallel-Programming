
#include <hip/hip_runtime.h>
// #include <cmath>
// #include <iostream>
// #include <cuda_fp16.h>
// #include "gpu-new-forward.h"

// OP1 for FP16 arithmetic


// void checkCudaErrors(cudaError_t err) {
//     if (err != cudaSuccess) {
//         std::cerr << "CUDA error: " << cudaGetErrorString(err) << std::endl;
//         exit(EXIT_FAILURE);
//     }
// }
// // op1 FP16 arithmetic
// __global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S) {
//     const int H_out = (H - K) / S + 1;
//     const int W_out = (W - K) / S + 1;

//     int w = blockIdx.x * blockDim.x + threadIdx.x;
//     int h = blockIdx.y * blockDim.y + threadIdx.y;
//     int m = blockIdx.z; 

//     if (w < W_out && h < H_out && m < M) {
//         for (int b = 0; b < B; b++) {
//             half accum = __float2half(0.0f);
//             for (int c = 0; c < C; c++) { 
//                 for (int p = 0; p < K; p++) { 
//                     for (int q = 0; q < K; q++) {
//                         int h_in = h * S + p;
//                         int w_in = w * S + q;
//                         half input_val = __float2half(input[b * (C * H * W) + c * (H * W) + h_in * W + w_in]);
//                         half mask_val = __float2half(mask[m * (C * K * K) + c * (K * K) + p * K + q]);
//                         accum = __hadd(accum, __hmul(input_val, mask_val));
//                     }
//                 }
//             }
//             output[b * (M * H_out * W_out) + m * (H_out * W_out) + h * W_out + w] = __half2float(accum);
//         }
//     }
// }
	
// __host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
// {
//     cudaError_t err;
//     size_t input_size = B * C * H * W * sizeof(float);
//     size_t output_size = B * M * ((H - K) / S + 1) * ((W - K) / S + 1) * sizeof(float);
//     size_t mask_size = M * C * K * K * sizeof(float);

//     err = cudaMalloc((void**)device_input_ptr, input_size);
//     checkCudaErrors(err);
//     err = cudaMalloc((void**)device_output_ptr, output_size);
//     checkCudaErrors(err);
//     err = cudaMalloc((void**)device_mask_ptr, mask_size);
//     checkCudaErrors(err);

//     err = cudaMemcpy(*device_input_ptr, host_input, input_size, cudaMemcpyHostToDevice);
//     checkCudaErrors(err);
//     err = cudaMemcpy(*device_mask_ptr, host_mask, mask_size, cudaMemcpyHostToDevice);
//     checkCudaErrors(err);
//     err = cudaMemset(*device_output_ptr, 0, output_size);
//     checkCudaErrors(err);
// }

// __host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
// {
//     const int H_out = (H - K) / S + 1;
//     const int W_out = (W - K) / S + 1;
    
//     dim3 blockDim(16, 16);
//     int gridW = (W_out + blockDim.x - 1) / blockDim.x;
//     int gridH = (H_out + blockDim.y - 1) / blockDim.y;

//     dim3 gridDim(gridW, gridH, M); 
//     conv_forward_kernel<<<gridDim, blockDim>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    
//     checkCudaErrors(cudaGetLastError());
//     checkCudaErrors(cudaDeviceSynchronize());

// }



// __host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
// {
//     // Copy the output back to host
//     const int H_out = (H - K) / S + 1;
//     const int W_out = (W - K) / S + 1;

//     size_t output_size = B * M * H_out * W_out * sizeof(float);
//     checkCudaErrors(cudaMemcpy(host_output, device_output, output_size, cudaMemcpyDeviceToHost));
   
//     checkCudaErrors(cudaFree(device_input));
//     checkCudaErrors(cudaFree(device_output));
//     checkCudaErrors(cudaFree(device_mask));

// }



